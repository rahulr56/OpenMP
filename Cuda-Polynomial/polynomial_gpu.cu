#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>

__global__
void computePolynomial(float*array, float* poly, int degree, int n)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < n )
  {
    float result = 0.0;
    float xtothepowerof = 1.0;
    for ( int x = 0; x < degree; ++x)
    {
      result += xtothepowerof * poly[x];
      xtothepowerof *= array[i];
    }
    array[i] = result;
  }
}
__global__
void polynomial_expansion (float* poly, int degree,
			   int n, float* array) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if( i < n )
  {
    float result = 0.0;
    float xtothepowerof = 1.0;
    for ( int x = 0; x < degree; ++x)
    {
      result += xtothepowerof * poly[x];
      xtothepowerof *= array[i];
    }
    array[i] = result;
  }
}

int main (int argc, char* argv[]) 
{
  if (argc < 3) 
  {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); 
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  float *gpuArray, *gpuPoly;

  hipMallocManaged(&gpuArray, n*sizeof(float));
  hipMallocManaged(&gpuPoly, (degree+1)*sizeof(float));
  
  cudaMemCpy(gpuArray, array, n*sizeof(float), hipMemcpyHostToDevice);
  cudaMemCpy(gpuPoly, poly, n*sizeof(float), hipMemcpyHostToDevice);

  polynomial_expansion<<<(n+255)/256, 256>>>(gpuPoly, degree, n, gpuArray);
  cudaMemCpy(array, gpuArray, n*sizeof(float), hipMemcpyDeviceToHost);

  hipFree(gpuArray);
  hipFree(gpuPoly);

  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();
  
  for (int iter = 0; iter<nbiter; ++iter)
    polynomial_expansion (poly, degree, n, array);
  cudaDeviceSychronze();
  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;

  return 0;
}
